
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10

using namespace std;

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void **) &dev_a, N * sizeof(int));
    hipMalloc((void **) &dev_b, N * sizeof(int));
    hipMalloc((void **) &dev_c, N * sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * 1;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyDeviceToDevice);

    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToDevice);

    // 显示结果
    for (int i = 0; i < N; i++) {
        cout << "a[i]+b[i]+c[i]=" << a[i] + b[i] + c[i] << endl;
    }

    // 释放内存（todo：不释放会如何）
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
